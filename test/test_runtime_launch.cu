#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "test_utils.h"
#include <unistd.h>


__global__ void add(float* a, float* b, float* c) {
    int idx = threadIdx.x;
    c[idx] = a[idx] + b[idx];
}

__global__ void computeKernel(double* data, int N, int iterations) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N) {
        double temp = 0.0;
        temp += sin(data[tid]) * cos(data[tid]);
        data[tid] = temp;
    }
}

int main() {
    float *a, *b, *c;
    CHECK_RUNTIME_API(hipMalloc(&a, 1024 * sizeof(float)));
    CHECK_RUNTIME_API(hipMalloc(&b, 1024 * sizeof(float)));
    CHECK_RUNTIME_API(hipMalloc(&c, 1024 * sizeof(float)));

    add<<<1, 1024>>>(a, b, c);

    int N = 1 << 27; 
    double* d_data;

    hipMalloc(&d_data, N * sizeof(double));

    int threadsPerBlock = 256;
    int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;

    int iterations = 1000000; 
    int num_launches = 100; 

    for (int i = 0; i < num_launches; ++i) {
        computeKernel<<<blocks, threadsPerBlock>>>(d_data, N, iterations);
        hipDeviceSynchronize();  
    }

    hipFree(d_data);

    sleep(100);
    printf("completed");
    return 0;
}
